#include "hip/hip_runtime.h"
 //Compiler command-line:
 //$ nvcc -lineinfo rnd_test.cu -o rnd_test
 //Profiler command-line:
 //$ nvvp ./rnd_test

//Includes the mtrand methods
extern "C" {
//Comment out this #define if you want to run on device!
//#define HOST
#include "mtrand.c"
}

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <iterator>
#include <vector>
#include <string>
#include <chrono>

#ifndef HOST
#define CUDA_GLOBAL __global__
//If HOST is not defined, then this code is running on the GPU (device)
#else
#define CUDA_GLOBAL
#endif

//Computes the random vals and stores it in the array
CUDA_GLOBAL void kernel(double *rndVals, int n){
    //Creates rndGen structure
    struct MTrand_Info rndGen;

#ifndef HOST
    //Following the logic from NVIDIA's beginners tutorial
    
    //Index is the thread's offset from the beginning of the block and the thread's index added in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //Initializes MTrand base call and changes seed each time based upon threadId
    MTrand_init(&rndGen, 5489+threadIdx.x);
    //Checks to make sure the index is within the size of the array
    if (index < n) {
        rndVals[index] = MTrand_get(&rndGen);
    }
#else
    for (int i = 0; i < n; i++) {
        //Initialize MTrand base call and changes seed each time based upon index
        MTrand_init(&rndGen, 5489 + i);
        rndVals[i] = MTrand_get(&rndGen);
    }
#endif
}

//Calls the kernel/function
void genRndNums(const int reps = 100) {

    //Create a variable to hold all of the random numbers
    double *rndVals;
    
    //Allocate variable in unified memory
    //(memory available to both CPU and GPU)
    hipMallocManaged(&rndVals, reps * sizeof(double));

#ifndef HOST
    //Sets up the block size and number of blocks for the kernel

    //Included in order to get the maximum from each streaming multiprocessor
    //inside of GPU
    int blockSize = 1024;
    //Calculates the number of thread blocks in the grid
    int numOfBlocks = (int) ceil( (float) reps / blockSize);

    //Calls the CUDA kernel and generates 'reps' number of random numbers
    kernel<<<numOfBlocks, blockSize>>>(rndVals, reps);
#else
    //Calls the function normally and generates 'reps' number of random numbers
    kernel(rndVals, reps);
#endif

    //Waits for the GPU code to finish executing before re-accessing the host
    //Synchronizes devices
    hipDeviceSynchronize();
    
    //Loop through rndVals and prints out the results
    for (int i = 0; (i < reps); i++) {
        std::cout << rndVals[i] << " ";
    }
    std::cout << std::endl;
    
    //Free Memory
    hipFree(rndVals);

}

//Calls getRndNums
//Defaults to 100 reps if arguement is not supplied
int main(int argc, char *argv[]) {
    auto startTime = std::chrono::high_resolution_clock::now();
    const int reps = (argc > 1 ? std::stoi( argv[1] ) : 100);
    genRndNums(reps);
    std::cout << "Success, " << reps << " random numbers created!" << std::endl;
#ifdef HOST
    std::cout << "Compiled on HOST (CPU)" << std::endl;
#else
    std::cout << "Compiled on DEVICE (GPU)" << std::endl;
#endif
    auto stopTime = std::chrono::high_resolution_clock::now();

    //Calculates the time take for the code to run
    auto time = std::chrono::duration_cast<std::chrono::microseconds>(stopTime - startTime);

    std::cout << "Time Taken: " <<
        time.count() << " microseconds" << std::endl;
    return 0;
}
